#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>B
#include <stdint.h>
#include <assert.h>
#include <random>
#include <chrono>

// AVX headers
#include <immintrin.h>
#include <x86intrin.h>

#include "iec_units.h"

using namespace std::chrono;


// Choose datatype
//----------------
// #define USE_FLOAT32
#define USE_FLOAT64
// #define USE_INT32
// #define USE_INT64

// Choose array size
//------------------
#define ARRAY_SIZE  _256MiB


#ifdef USE_FLOAT32
    typedef float T;
    #define T_print(val) printf("%f ", (val))
#endif
#ifdef USE_FLOAT64
    typedef double T;
    #define T_print(val) printf("%f ", (val))
#endif
#ifdef USE_INT32
    typedef int32_t T;
    #define T_print(val) printf("%3d ", (val))
#endif
#ifndef USE_FLOAT32
    #ifndef USE_FLOAT64
        #ifndef USE_INT32
            typedef int64_t T;
            #define T_print(val) printf("%3ld ", (val))
            #define USE_INT64
        #endif
    #endif
#endif


#define NxN         (ARRAY_SIZE / sizeof(T))
#define N           ((size_t) floor(sqrt(NxN)))
#define ACTUAL_SIZE (N*N * sizeof(T))

// #define N           1024
// #define NxN         (N*N)
// #define ARRAY_SIZE  (NxN * sizeof(T))
// #define ACTUAL_SIZE ARRAY_SIZE


#define STR_BUFF_OFFSET     16
#define TIME_STR_WIDTH      12




__global__ void gpu_matmul(T* A, T* B, T* C);
__global__ void gpu_matmul_trans(T* A, T* B, T* C);

inline void cpu_matmul(T* A, T* B, T* C);
inline void cpu_matmul_trans(T* A, T* B, T* C);
inline void avx256_matmul_int32(T* A, T* B, T* C);

void allocate(T** ptr, size_t n);
void print(T* A, T* B, T* C);




int main() {

    assert(ARRAY_SIZE >= ACTUAL_SIZE);

    // Set locale for printf
    setlocale(LC_NUMERIC, "");
    // String buffer
    char  str_buff[64];
    char* __str_buff = str_buff + STR_BUFF_OFFSET;
    for (size_t i=0; i<STR_BUFF_OFFSET; i++) {
        str_buff[i] = ' ';
    }

    // Query GPU device properties
    int deviceId;
    hipGetDevice(&deviceId);
    // hipDeviceProp_t props;
    // hipGetDeviceProperties(&props, deviceId);

    // int computeCapabilityMajor = props.major;
    // int computeCapabilityMinor = props.minor;
    // int multiProcessorCount = props.multiProcessorCount;
    // int warpSize = props.warpSize;

    // printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n\n\n",
    //         deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
    // printf("Number of SMs: %d\nWarp size: %d\n\n", multiProcessorCount, warpSize);


    // Allocate memory
    T* A;
    T* B;
    T* C;
    T* BASELINE;
    allocate(&A, ARRAY_SIZE);
    allocate(&B, ARRAY_SIZE);
    allocate(&C, ARRAY_SIZE);
    allocate(&BASELINE, ARRAY_SIZE);

    #ifdef USE_FLOAT32
        printf("Using single-precision 32-bit floating point");
    #endif
    #ifdef USE_FLOAT64
        printf("Using double-precision 64-bit floating point");
    #endif
    #ifdef USE_INT32
        printf("Using 32-bit integer");
    #endif
    #ifdef USE_INT64
        printf("Using 64-bit integer");
    #endif
    format_iec(str_buff, ACTUAL_SIZE);
    printf(" with array size %s\n", str_buff);


    // Initialize
    // printf("Initializing...");
    fflush(stdout);
    hipMemPrefetchAsync(A, ARRAY_SIZE, hipCpuDeviceId);
    hipMemPrefetchAsync(B, ARRAY_SIZE, hipCpuDeviceId);
    hipMemPrefetchAsync(C, ARRAY_SIZE, hipCpuDeviceId);
    hipMemPrefetchAsync(BASELINE, ARRAY_SIZE, hipCpuDeviceId);
    srand(0);
    #ifdef USE_FLOAT32
        std::uniform_real_distribution<T> my_rand(-1,1);
        std::default_random_engine rand_engine;
    #endif
    for (size_t i=0; i<NxN; i++) {
        #ifdef USE_FLOAT32
            A[i] = my_rand(rand_engine);
            B[i] = my_rand(rand_engine);
        #else
            if (N <= 8) {
                // Easier to read/debug
                A[i] = rand() % 10;
                B[i] = rand() % 10;
            } else {
                A[i] = rand();
                B[i] = rand();
            }
        #endif
        C[i] = -1;
        BASELINE[i] = -1;
    }
    // printf("done\n");

    // CPU
    printf("CPU...      ");
    fflush(stdout);
    auto start = high_resolution_clock::now();
    cpu_matmul(A, B, BASELINE);
    auto stop = high_resolution_clock::now();
    auto time = duration_cast<microseconds>(stop - start);
    sprintf(__str_buff, "%'ld", time.count());
    char* time_str = __str_buff - (TIME_STR_WIDTH - strlen(__str_buff));
    printf("done  %s us\n", time_str);

    // CPU Transposed
    printf("CPUt...     ");
    fflush(stdout);
    start = high_resolution_clock::now();
    cpu_matmul_trans(A, B, C);
    stop = high_resolution_clock::now();
    time = duration_cast<microseconds>(stop - start);
    sprintf(__str_buff, "%'ld", time.count());
    time_str = __str_buff - (TIME_STR_WIDTH - strlen(__str_buff));
    printf("done  %s us\n", time_str);


    // GPU
    printf("GPU...      ");
    fflush(stdout);
    hipMemPrefetchAsync(A, ARRAY_SIZE, deviceId);
    hipMemPrefetchAsync(B, ARRAY_SIZE, deviceId);
    hipMemPrefetchAsync(C, ARRAY_SIZE, deviceId);
    start = high_resolution_clock::now();
    gpu_matmul<<<dim3(N,N),1>>>(A, B, C);
    hipDeviceSynchronize();
    stop = high_resolution_clock::now();
    time = duration_cast<microseconds>(stop - start);
    sprintf(__str_buff, "%'ld", time.count());
    time_str = __str_buff - (TIME_STR_WIDTH - strlen(__str_buff));
    printf("done  %s us\n", time_str);


    // GPU Transposed
    printf("GPUt...     ");
    fflush(stdout);
    hipMemPrefetchAsync(A, ARRAY_SIZE, deviceId);
    hipMemPrefetchAsync(B, ARRAY_SIZE, deviceId);
    hipMemPrefetchAsync(C, ARRAY_SIZE, deviceId);
    start = high_resolution_clock::now();
    gpu_matmul_trans<<<dim3(N,N),1>>>(A, B, C);
    hipDeviceSynchronize();
    stop = high_resolution_clock::now();
    time = duration_cast<microseconds>(stop - start);
    sprintf(__str_buff, "%'ld", time.count());
    time_str = __str_buff - (TIME_STR_WIDTH - strlen(__str_buff));
    printf("done  %s us\n", time_str);

    // AVX
    #ifdef USE_INT32
    if (ARRAY_SIZE == ACTUAL_SIZE) {
        hipMemPrefetchAsync(A, ARRAY_SIZE, hipCpuDeviceId);
        hipMemPrefetchAsync(B, ARRAY_SIZE, hipCpuDeviceId);
        hipMemPrefetchAsync(C, ARRAY_SIZE, hipCpuDeviceId);
        printf("AVX-256...  ");
        fflush(stdout);
        // Requires C to be initialized to 0
        for (size_t i=0; i<NxN; i++) {
            C[i] = 0;
        }
        start = high_resolution_clock::now();
        avx256_matmul_int32(A, B, C);
        stop = high_resolution_clock::now();
        time = duration_cast<microseconds>(stop - start);
        sprintf(__str_buff, "%'ld", time.count());
        time_str = __str_buff - (TIME_STR_WIDTH - strlen(__str_buff));
        printf("done  %s us\n", time_str);
        
        // Verify
        printf("Verifying AVX-256 result...");
        fflush(stdout);
        // hipMemPrefetchAsync(A, ARRAY_SIZE, hipCpuDeviceId);
        // hipMemPrefetchAsync(B, ARRAY_SIZE, hipCpuDeviceId);
        // hipMemPrefetchAsync(C, ARRAY_SIZE, hipCpuDeviceId);
        bool valid = true;
        for (size_t row=0; row<N; row++) {
            for (size_t col=0; col<N; col++) {
                if (C[row*N + col] != BASELINE[row*N + col]) {
                    valid = false;
                }
            }
        }
        if (!valid) {
            printf("\n\n!!!ERROR: CPU and AVX-256 results do not match.\n\n");
            if (N <= 16) {
                printf("\nCPU:\n");
                print(A, B, BASELINE);
            }
            if (N <= 16) {
                printf("\n\nAVX-256:\n");
                print(A, B, C);
            }
        } else {
            printf("done\n");
        }
    } else {
        printf("AVX-256 matrix dimension must be power of 2. Currently N=%ld\n", N);
    }
    #else
        printf("AVX-256 only used with int32\n");
    #endif

}


__global__ void gpu_matmul(T* A, T* B, T* C) {
    T sum = 0;
    for (size_t k=0; k<N; k++) {
        sum += A[blockIdx.x*N + k] * B[k*N + blockIdx.y]; // B not transposed
    }
    C[blockIdx.x*N + blockIdx.y] = sum;
}


__global__ void gpu_matmul_trans(T* A, T* B, T* C) {
    T sum = 0;
    for (size_t k=0; k<N; k++) {
        sum += A[blockIdx.x*N + k] * B[k + N*blockIdx.y]; // B transposed
    }
    C[blockIdx.x*N + blockIdx.y] = sum;
}


inline void cpu_matmul(T* A, T* B, T* C) {
    for (size_t row=0; row<N; row++) {
        for (size_t col=0; col<N; col++) {
            C[row*N+col] = 0.0;
            for (size_t k=0; k<N; k++) {
                C[row*N+col] += A[row*N+k] * B[k*N+col];
            }
        }
    }
}


inline void cpu_matmul_trans(T* A, T* B, T* C) {
    for (size_t row=0; row<N; row++) {
        for (size_t col=0; col<N; col++) {
            C[row*N+col] = 0.0;
            for (size_t k=0; k<N; k++) {
                C[row*N+col] += A[row*N+k] * B[k+N*col];
            }
        }
    }
}


// https://codereview.stackexchange.com/questions/177616/avx-simd-in-matrix-multiplication
inline void avx256_matmul_int32(T* A, T* B, T* C) {
    __m256i vec_multi_res = _mm256_setzero_si256(); //Initialize vector to zero
    __m256i vec_A = _mm256_setzero_si256();         //Initialize vector to zero
    __m256i vec_B = _mm256_setzero_si256();         //Initialize vector to zero

    size_t i, j, k;
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; ++j)
        {
            //Stores one element in A and use it in all computations needed before proceeding
            //Stores as vector to increase computations per cycle
            vec_A = _mm256_set1_epi32(A[i*N + j]);

            for (k = 0; k < N; k += 8)
            {
                vec_B = _mm256_loadu_si256((__m256i*)&B[j*N + k]);         //Stores row of second matrix (eight in each iteration)
                vec_multi_res = _mm256_loadu_si256((__m256i*)&C[i*N + k]); //Loads the C matrix row as a vector
                vec_multi_res = _mm256_add_epi32(vec_multi_res ,_mm256_mullo_epi32(vec_A, vec_B)); //Multiplies the vectors and adds to the C vector

                _mm256_storeu_si256((__m256i*)&C[i*N + k], vec_multi_res); //Stores the C vector into the C array
            }
        }
    }
}


// https://www.intel.com/content/www/us/en/developer/articles/technical/accelerating-compute-intensive-workloads-with-intel-avx-512-using-microsoft-visual-studio.html
inline void avx_matmul_intel(T* A, T* B, T* C) {

}


void allocate(T** ptr, size_t size) {
    hipError_t err = hipMallocManaged(ptr, size);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        assert(NULL != *ptr);
    }
}


void print(T* A, T* B, T* C) {
    for (size_t row=0; row<N; row++) {
        printf("| ");
        for (size_t col=0; col<N; col++) {
            // printf("%d ", A[row*N+col]);
            T_print(A[row*N+col]);
        }
        printf("|  | ");
        for (size_t col=0; col<N; col++) {
            // printf("%d ", B[row*N+col]);
            T_print(B[row*N+col]);
        }
        printf("|  =  | ");
        for (size_t col=0; col<N; col++) {
            // printf("%d ", C[row*N+col]);
            T_print(C[row*N+col]);
        }
        printf("|\n");
    }
    printf("\n");
}
